
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#define BLOCK_SIZE 16

__global__ 
void MultiMatrices(float *d_ma, float *d_mb, float *d_mp, int Width, int m, int n){
    
    int Row = blockIdx.y*blockDim.y+threadIdx.y;

    int Col = blockIdx.x*blockDim.x+threadIdx.x;

    if ((Row < m) && (Col < n)) { 
        float Pvalue = 0;
        
        for (int k = 0; k < Width; k++) {
            Pvalue += d_ma[Row*Width+k]*d_mb[k*n+Col];
        }   
        d_mp[Row*n+Col] = Pvalue;
    }

}

void MM_cpu(float *h_ma, float *h_mb, float *h_mp, int Width, int m, int n) {
    float Pvalue = 0;

    for(int Row = 0; Row < m; Row++) {
        for(int Col = 0; Col < n; Col++) {
            
            for (int k = 0; k < Width; k++) {
                Pvalue += h_ma[Row*Width+k]*h_mb[k*n+Col];
            }
            h_mp[Row*n+Col] = Pvalue;
            Pvalue = 0;
        }
    }
} 

void llenarMatriz(float *matrix, int x, int y, float v){
    for (int i = 0; i < x; i++) {       
        for (int j = 0; j < y; j++) {
            matrix[(i*y)+j] = v;
        }
    }
}


void imprimirMatriz(float *matrix, int x, int y){
    for (int i = 0; i < x; i++) {
        for (int j = 0; j < y; j++) {
            printf("%f ", matrix[(i*y)+j]);
        }
        printf("\n");
    }
}


int main(int argc, char *argv[]){
    //Programa <archivo>

    if ( argc != 2 ) {
        //Salir del programa
        printf("Fallo al ingresar el argumento\n");
    }
    else 
    {
        FILE *fp;
        float floatBuffer;

        int ma, ka, kb, nb;
        int m, k, n;
    
        float *h_a, *h_b, *h_c;
        float *d_a, *d_b, *d_c;

        fp = fopen (argv[1], "r");
        if (fp == NULL) {
            perror ("Error al abrir el archivo");
        }
        else
        {
                     
            fscanf(fp, "%d", &ma);
            fscanf(fp, "%d", &ka);

            hipHostMalloc((void **) &h_a, sizeof(float)*ma*ka, hipHostMallocDefault);

            for (int i = 0; i < ma; i++) {
                for (int j = 0; j < ka; j++) {
                    fscanf(fp, "%f", &floatBuffer);
                    h_a[(i*ka) + j] = floatBuffer;
                }
            }
                
            fscanf(fp, "%d", &kb);
            fscanf(fp, "%d", &nb);

            hipHostMalloc((void **) &h_b, sizeof(float)*kb*nb, hipHostMallocDefault);

            for (int i = 0; i < kb; i++) {
                for (int j = 0; j < nb; j++) {
                    fscanf(fp, "%f", &floatBuffer);
                    h_b[(i*nb) + j] = floatBuffer;
                }
            }
            
            fclose(fp);
        }

        if (ka != kb) {
            printf("la matriz no cumple con la condicion de multiplicatividad");
            return 0;
        }

        m = ma;
        k = ka;
        n = nb;               

        hipHostMalloc((void **) &h_c, sizeof(float)*m*n, hipHostMallocDefault);
        llenarMatriz(h_c, m, n, 0);
                
        imprimirMatriz(h_a, m, k);
        imprimirMatriz(h_b, k, n);
        imprimirMatriz(h_c, m, n);
        
        //MM_cpu(h_a, h_b, h_c, k, m, n);
        
        //no hay que olvidarse de declarar espacio en el device
        hipMalloc((void **) &d_a, sizeof(float)*m*k);
        hipMalloc((void **) &d_b, sizeof(float)*k*n);
        hipMalloc((void **) &d_c, sizeof(float)*m*n);
        
        

        hipMemcpy(d_a, h_a, sizeof(float)*m*k, hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, sizeof(float)*k*n, hipMemcpyHostToDevice);
        hipMemcpy(d_c, h_c, sizeof(float)*m*n, hipMemcpyHostToDevice);

                

        unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
        unsigned int grid_cols = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

        dim3 dimGrid(grid_cols, grid_rows);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
   
        hipError_t MMErr;
        hipError_t asyncErr;

        
    
        MultiMatrices<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, k, m, n);

        hipMemcpy(h_c, d_c, sizeof(float)*m*n, hipMemcpyDeviceToHost);        

        MMErr = hipGetLastError();
        if(MMErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(MMErr));

        asyncErr = hipDeviceSynchronize();
        if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));
        

        imprimirMatriz(h_c, m, n);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);

        hipHostFree(h_a);
        hipHostFree(h_b);
        hipHostFree(h_c);    

    }

    return 0;
}
